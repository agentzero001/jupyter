
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_cuda() {
    printf("hello Cuda \n");
    printf("blockIdx.x: %d, blockIdx.y: %d, threadidx.x: %d, threadIdx.y: %d, threadidx.z: %d\n",
    blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z);

}

int main(int argc, char **argv) {
    hello_cuda<<<3, 4>>>();
    hipDeviceSynchronize();
    return 0;
}